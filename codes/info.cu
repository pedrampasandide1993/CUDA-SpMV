// nvcc -o info info.cu
// ./info
#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
    int device_id = 0;
    hipSetDevice(device_id);

    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device_id);
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device_id);

    printf("Compute capability of device %d: %d.%d\n", device_id, major, minor);

    int shared_mem_per_block;
    hipDeviceGetAttribute(&shared_mem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id);
    printf("Shared memory per block of device %d: %d bytes\n", device_id, shared_mem_per_block);

    int regs_per_block;
    hipDeviceGetAttribute(&regs_per_block, hipDeviceAttributeMaxRegistersPerBlock, device_id);
    printf("Registers per block of device %d: %d\n", device_id, regs_per_block);

    int warp_size;
    hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, device_id);
    printf("Warp size of device %d: %d threads\n", device_id, warp_size);

    int max_threads_per_block;
    hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, device_id);
    printf("Max threads per block of device %d: %d threads\n", device_id, max_threads_per_block);

    int max_block_dim_x, max_block_dim_y, max_block_dim_z;
    hipDeviceGetAttribute(&max_block_dim_x, hipDeviceAttributeMaxBlockDimX, device_id);
    hipDeviceGetAttribute(&max_block_dim_y, hipDeviceAttributeMaxBlockDimY, device_id);
    hipDeviceGetAttribute(&max_block_dim_z, hipDeviceAttributeMaxBlockDimZ, device_id);
    printf("Max thread dimensions of device %d: %d x %d x %d\n", device_id, max_block_dim_x, max_block_dim_y, max_block_dim_z);

    int total_constant_mem;
    hipDeviceGetAttribute(&total_constant_mem, hipDeviceAttributeTotalConstantMemory, device_id);
    printf("Total constant memory of device %d: %d bytes\n", device_id, total_constant_mem);

    int mp_count;
    hipDeviceGetAttribute(&mp_count, hipDeviceAttributeMultiprocessorCount, device_id);
    printf("Multiprocessor count of device %d: %d\n", device_id, mp_count);

    int memory_bus_width;
    hipDeviceGetAttribute(&memory_bus_width, hipDeviceAttributeMemoryBusWidth, device_id);
    printf("Memory bus width of device %d: %d bits\n", device_id, memory_bus_width);

    int l2_cache_size;
    hipDeviceGetAttribute(&l2_cache_size, hipDeviceAttributeL2CacheSize, device_id);
    printf("L2 cache size of device %d: %d bytes\n", device_id, l2_cache_size);

    int max_threads_per_sm;
    hipDeviceGetAttribute(&max_threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, device_id);
    printf("Max threads per SM of device %d: %d\n", device_id, max_threads_per_sm);

    return 0;
}